#include "hip/hip_runtime.h"
//in place 1d dst


#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include "hip/device_functions.h"
#include "transformfunc.h"


# define M_PI   3.14159265358979323846


// #define TIME_TEST



hipfftHandle plan_dst2_nocubic;

#ifdef TIME_TEST
static long long time_preOp, time_postOp,time_trans_xzy,time_trans_zyx,time_cufft;



static void timeBegin(struct timeval *tBegin){
	gettimeofday(tBegin, NULL);
}

static long long timeEnd(struct timeval tBegin){
 
	 struct timeval tEnd;
     gettimeofday(&tEnd, NULL);
   
     long long usec=(tEnd.tv_sec-tBegin.tv_sec)*1000*1000+tEnd.tv_usec-tBegin.tv_usec;
	
	 return usec;
}
#endif



__global__ void preOp_dst2_inplace(double* in,int N,int batch){
    int itx=threadIdx.x;
    int ibx=blockIdx.x;
    int iby=blockIdx.y;
    double* pin=in+iby*(N+2)*(batch+2)+ibx*(N+2);
    extern __shared__ double sh_in[];

    if(itx<N/2){
        sh_in[itx]=pin[itx+1];
        sh_in[itx+N/2]=pin[itx+N/2+1];
        
        
    }
    __syncthreads();
    if(itx<N/2+1){
        if(itx==0){
            pin[0]=sh_in[0];
            pin[1]=0;
        }else if(itx==N/2){
            pin[N]=-sh_in[N-1];
            pin[N+1]=0;
        }else{
            pin[itx*2]=(sh_in[itx*2]-sh_in[itx*2-1])/2;
            pin[itx*2+1]=-((sh_in[itx*2]+sh_in[itx*2-1])/2);
        }
    }
    

}
__global__ void postOp_dst2_inplace(double* in,int N,int batch){
    int itx=threadIdx.x;
    int ibx=blockIdx.x;
    int iby=blockIdx.y;
    double* pin=in+iby*(N+2)*(batch+2)+ibx*(N+2);
    extern __shared__ double sh_in[];
    if(itx<N/2){
        sh_in[itx]=pin[itx];
        sh_in[itx+N/2]=pin[itx+N/2];
    }
    __syncthreads();

    if(itx<N/2+1){
        if(itx!=0){
            double sina;
            double cosa;
            sincos((itx*M_PI/(2*N)),&sina,&cosa);
            double Ta=sh_in[itx]+sh_in[N-itx];
            double Tb=sh_in[itx]-sh_in[N-itx];
            // double sina=sin(itx*M_PI/(2*N));
            // double cosa=cos(itx*M_PI/(2*N));

            pin[itx]=(Ta*sina+Tb*cosa)/2;
            pin[N-itx]=(Ta*cosa-Tb*sina)/2;
        }else{
            pin[0]=0;
            pin[N]=sh_in[0];
        }
    }
    

}




void dofft_dst2_inplace(double *d_data , int DATA_SIZE,int batch,int nLayer){
    int n[1]={DATA_SIZE};
    int inembeb[1]={(DATA_SIZE+2)/2};
    int onembeb[1]={(DATA_SIZE+2)};
    hipfftResult r = hipfftPlanMany(&plan_dst2_nocubic,1,n,
                    inembeb,1,(DATA_SIZE+2)/2,
                    onembeb,1,(DATA_SIZE+2),
                    HIPFFT_Z2D, (batch+2)*(nLayer+2));

    if(r!=0){
        printf("CUFFT FAILED! ERROR CODE: %s\n",cufftresultcode[r]);
        exit(0);
    }
    
    hipfftExecZ2D(plan_dst2_nocubic, reinterpret_cast<hipfftDoubleComplex *>(d_data),
                                reinterpret_cast<double *>(d_data));
  



}


//dst2也默认输入和输出数组的第一位为0，与dst3一致
void run_3d_dst_2_inplace_nocubic(double *d_data , int DATA_SIZE, int batch ,int nLayer){
    

    #ifdef TIME_TEST
    time_postOp=0;
    time_preOp=0;
    time_trans_xzy=0;
    time_trans_zyx=0;
    time_cufft=0;
    #endif
    

    dim3 preOpGridDim;
    preOpGridDim.x=batch+1;
    preOpGridDim.y=nLayer+1;
    preOpGridDim.z=1;

    #ifdef TIME_TEST
    struct timeval tBegin1;
    timeBegin(&tBegin1);
    #endif

    preOp_dst2_inplace<<<preOpGridDim,DATA_SIZE/2+1,sizeof(double)*DATA_SIZE>>>(d_data,DATA_SIZE,batch);
   
    #ifdef TIME_TEST
    hipDeviceSynchronize();
    time_preOp = timeEnd(tBegin1);
    #endif 

    #ifdef TIME_TEST
    struct timeval tBegin2;
    timeBegin(&tBegin2);
    #endif

    dofft_dst2_inplace(d_data,DATA_SIZE,batch,nLayer);
    

    #ifdef TIME_TEST
    hipDeviceSynchronize();
    time_cufft += timeEnd(tBegin2);
    #endif 

    


    dim3 postOpGridDim;
    postOpGridDim.x=batch+1;
    postOpGridDim.y=nLayer+1;
    postOpGridDim.z=1;

    #ifdef TIME_TEST
    struct timeval tBegin3;
    timeBegin(&tBegin3);
    #endif

    postOp_dst2_inplace<<<postOpGridDim,DATA_SIZE/2+1,sizeof(double)*(DATA_SIZE)>>>(d_data,DATA_SIZE,batch);

    #ifdef TIME_TEST
    hipDeviceSynchronize();
    time_postOp += timeEnd(tBegin3);
    #endif 

    
    #ifdef TIME_TEST
    hipDeviceSynchronize();
    printf("timepreOp:    count=3  totaltime=%lld  avetime=%lld \n",time_preOp,time_preOp/3);
    printf("timepostOp:   count=3  totaltime=%lld  avetime=%lld \n",time_postOp,time_postOp/3);
    printf("timecufft:    count=3  totaltime=%lld  avetime=%lld \n",time_cufft,time_cufft/3);
    printf("timetransxzy: count=2  totaltime=%lld  avetime=%lld \n",time_trans_xzy,time_trans_xzy/2);
    printf("timetranszyx: count=2  totaltime=%lld  avetime=%lld \n",time_trans_zyx,time_trans_zyx/2);
    #endif

    



    
    freeMemory_dst2();
}


void run_3d_dst_2_inplace(double *d_data , int DATA_SIZE,hipfftHandle &plan_dst2_cubic){
    

    #ifdef TIME_TEST
    time_postOp=0;
    time_preOp=0;
    time_trans_xzy=0;
    time_trans_zyx=0;
    time_cufft=0;
    #endif
    

    dim3 preOpGridDim;
    preOpGridDim.x=DATA_SIZE+1;
    preOpGridDim.y=DATA_SIZE+1;
    preOpGridDim.z=1;

    #ifdef TIME_TEST
    struct timeval tBegin1;
    timeBegin(&tBegin1);
    #endif

    preOp_dst2_inplace<<<preOpGridDim,DATA_SIZE/2+1,sizeof(double)*DATA_SIZE>>>(d_data,DATA_SIZE,DATA_SIZE);
   
    #ifdef TIME_TEST
    hipDeviceSynchronize();
    time_preOp = timeEnd(tBegin1);
    #endif 
    
    #ifdef TIME_TEST
    struct timeval tBegin2;
    timeBegin(&tBegin2);
    #endif

    hipfftExecZ2D(plan_dst2_cubic, reinterpret_cast<hipfftDoubleComplex *>(d_data),
                                reinterpret_cast<double *>(d_data));

    #ifdef TIME_TEST
    hipDeviceSynchronize();
    time_cufft += timeEnd(tBegin2);
    #endif 

    


    dim3 postOpGridDim;
    postOpGridDim.x=DATA_SIZE+1;
    postOpGridDim.y=DATA_SIZE+1;
    postOpGridDim.z=1;

    #ifdef TIME_TEST
    struct timeval tBegin3;
    timeBegin(&tBegin3);
    #endif

    postOp_dst2_inplace<<<postOpGridDim,DATA_SIZE/2+1,sizeof(double)*(DATA_SIZE)>>>(d_data,DATA_SIZE,DATA_SIZE);

    #ifdef TIME_TEST
    hipDeviceSynchronize();
    time_postOp += timeEnd(tBegin3);
    #endif 


    #ifdef TIME_TEST
    hipDeviceSynchronize();
    printf("timepreOp:    count=3  totaltime=%lld  avetime=%lld \n",time_preOp,time_preOp/3);
    printf("timepostOp:   count=3  totaltime=%lld  avetime=%lld \n",time_postOp,time_postOp/3);
    printf("timecufft:    count=3  totaltime=%lld  avetime=%lld \n",time_cufft,time_cufft/3);
    printf("timetransxzy: count=2  totaltime=%lld  avetime=%lld \n",time_trans_xzy,time_trans_xzy/2);
    printf("timetranszyx: count=2  totaltime=%lld  avetime=%lld \n",time_trans_zyx,time_trans_zyx/2);
    #endif

    
}


//for nocubic
void freeMemory_dst2(){

    hipfftDestroy(plan_dst2_nocubic);


}
