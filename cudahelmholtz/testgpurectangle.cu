#include "hip/hip_runtime.h"
#include "hw3crt_wrapper.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include "time_.h"
#include "cuda_helmholtz_solver.h"
#include "ufunc.h"

#define PI 3.14159265358979323846
//#define DEBUG

using namespace std;

//in this code, we assume that solution domain is [0,1]^3
int main(int argc, char **argv)
{
	if (argc < 7)
	{
		printf("usage: ./testgpurectangle Nx Ny Nz xbc ybc zbc\n");
		exit(-1);
	}
	int Nx = atoi(argv[1]);
	int Ny = atoi(argv[2]);
	int Nz = atoi(argv[3]);
	int xbc = atoi(argv[4]);
	int ybc = atoi(argv[5]);
	int zbc = atoi(argv[6]);
	int nx=Nx; 
	int ny=Ny;
	int nz=Nz;
	double lambda = -1; // 
	int i, j, k;
	double dx = 1.0/nx;
	double dy = 1.0/ny;
	double dz = 1.0/nz;
	int sz = (nx+1)*(ny+1)*(nz+1);
	double *f =  new double[sz];
	double *bcl=NULL;
	double *bcr=NULL;
	double *bcb=NULL;
	double *bct=NULL;
	double *bcf=NULL;
	double *bce=NULL;

	if ( xbc == 3 || xbc == 4 )
		bcl = (double *) malloc(sizeof(double)*(ny+1)*(nz+1));
	if ( xbc == 3 || xbc == 2 )
		bcr = (double *) malloc(sizeof(double)*(ny+1)*(nz+1));
	if ( ybc == 3 || ybc == 4 )
		bcb = (double *) malloc(sizeof(double)*(nx+1)*(nz+1));
	if ( ybc == 3 || ybc == 2 )
		bct = (double *) malloc(sizeof(double)*(nx+1)*(nz+1));
	if ( zbc == 3 || zbc == 4 )
		bcf = (double *) malloc(sizeof(double)*(nx+1)*(ny+1));
	if ( zbc == 3 || zbc == 2 )
		bce = (double *) malloc(sizeof(double)*(nx+1)*(ny+1));

	//assign right hand side
	for (k=0; k<nz+1; k++)
		for (j=0; j<ny+1; j++)
			for (i=0; i<nx+1; i++)
				f[k*(nx+1)*(ny+1)+j*(nx+1)+i] = 
					laplaceufunc(i*dx,j*dy,k*dz)+lambda*ufunc(i*dx,j*dy,k*dz);

	//assign x boundary condition
	//for Dirichlet boundary condition
	for (k=0; k<nz+1; k++)
		for (j=0; j<ny+1; j++)
		{
			if ( xbc == 1 || xbc == 2 )
				f[k*(nx+1)*(ny+1)+j*(nx+1)+0] = ufunc(0*dx,j*dy,k*dz);
			if ( xbc == 1 || xbc == 4 )
				f[k*(nx+1)*(ny+1)+j*(nx+1)+nx] = ufunc(nx*dx,j*dy,k*dz);
		}
	//for Neumann boundary condition
	for (k=0; k<nz+1; k++)
		for (j=0; j<ny+1; j++)
		{
			if ( xbc == 3 || xbc == 4 )
				bcl[k*(ny+1)+j] = uxfunc(0*dx,j*dy,k*dz);
			if ( xbc == 3 || xbc == 2 )
				bcr[k*(ny+1)+j] = uxfunc(nx*dx,j*dy,k*dz);
		}

	//assign y boundary condition
	//for Dirichlet boundary condition
	for (k=0; k<nz+1; k++)
		for (i=0; i<nx+1; i++)
		{
			if ( ybc == 1 || ybc == 2 )
				f[k*(nx+1)*(ny+1)+0*(nx+1)+i] = ufunc(i*dx,0*dy,k*dz);
			if ( ybc == 1 || ybc == 4 )
				f[k*(nx+1)*(ny+1)+ny*(nx+1)+i] = ufunc(i*dx,ny*dy,k*dz);
		}
	//for Neumann boundary condition
	for (k=0; k<nz+1; k++)
		for (i=0; i<nx+1; i++)
		{
			if ( ybc == 3 || ybc == 4 )
				bcb[k*(nx+1)+i] = uyfunc(i*dx,0*dy,k*dz);
			if ( ybc == 3 || ybc == 2 )
				bct[k*(nx+1)+i] = uyfunc(i*dx,ny*dy,k*dz);
		}

	//assign z boundary condition
	//for Dirichlet boundary condition
	for (j=0; j<ny+1; j++)
		for (i=0; i<nx+1; i++)
		{
			if ( zbc == 1 || zbc == 2 )
				f[0*(nx+1)*(ny+1)+j*(nx+1)+i] = ufunc(i*dx,j*dy,0*dz);
			if ( zbc == 1 || zbc == 4 )
				f[nz*(nx+1)*(ny+1)+j*(nx+1)+i] = ufunc(i*dx,j*dy,nz*dz);
		}
	//for Neumann boundary condition
	for (j=0; j<ny+1; j++)
		for (i=0; i<nx+1; i++)
		{
			if ( zbc == 3 || zbc == 4 )
				bcf[j*(nx+1)+i] = uzfunc(i*dx,j*dy,0*dz);
			if ( zbc == 3 || zbc == 2 )
				bce[j*(nx+1)+i] = uzfunc(i*dx,j*dy,nz*dz);
		}

	double *f_d;
	double *bcl_d=NULL;
	double *bcr_d=NULL;
	double *bcb_d=NULL;
	double *bct_d=NULL;
	double *bcf_d=NULL;
	double *bce_d=NULL;

	hipMalloc(&f_d, sizeof(double)*sz);

	if ( xbc == 3 || xbc == 4 )
		hipMalloc(&bcl_d, sizeof(double)*(ny+1)*(nz+1));
	if ( xbc == 3 || xbc == 2 )
		hipMalloc(&bcr_d, sizeof(double)*(ny+1)*(nz+1));
	if ( ybc == 3 || ybc == 4 )
		hipMalloc(&bcb_d, sizeof(double)*(nx+1)*(nz+1));
	if ( ybc == 3 || ybc == 2 )
		hipMalloc(&bct_d, sizeof(double)*(nx+1)*(nz+1));
	if ( zbc == 3 || zbc == 4 )
		hipMalloc(&bcf_d, sizeof(double)*(nx+1)*(ny+1));
	if ( zbc == 3 || zbc == 2 )
		hipMalloc(&bce_d, sizeof(double)*(nx+1)*(ny+1));

	hipMemcpy(f_d, f, sizeof(double)*sz, hipMemcpyHostToDevice);

	if ( xbc == 3 || xbc == 4 )
		hipMemcpy(bcl_d, bcl, sizeof(double)*(ny+1)*(nz+1), hipMemcpyHostToDevice);
	if ( xbc == 3 || xbc == 2 )
		hipMemcpy(bcr_d, bcr, sizeof(double)*(ny+1)*(nz+1), hipMemcpyHostToDevice);
	if ( ybc == 3 || ybc == 4 )
		hipMemcpy(bcb_d, bcb, sizeof(double)*(nx+1)*(nz+1), hipMemcpyHostToDevice);
	if ( ybc == 3 || ybc == 2 )
		hipMemcpy(bct_d, bct, sizeof(double)*(nx+1)*(nz+1), hipMemcpyHostToDevice);
	if ( zbc == 3 || zbc == 4 )
		hipMemcpy(bcf_d, bcf, sizeof(double)*(nx+1)*(ny+1), hipMemcpyHostToDevice);
	if ( zbc == 3 || zbc == 2 )
		hipMemcpy(bce_d, bce, sizeof(double)*(nx+1)*(ny+1), hipMemcpyHostToDevice);

	//warm up
	cuda_helmholtz_solver(0, 1, nx, xbc, bcl_d, bcr_d, 
			  0, 1, ny, ybc, bcb_d, bct_d,  
			  0, 1, nz, zbc, bcf_d, bce_d, 
			  lambda, f_d);

	hipMemcpy(f_d, f, sizeof(double)*sz, hipMemcpyHostToDevice);

	//start timing
	time_(
	cuda_helmholtz_solver(0, 1, nx, xbc, bcl_d, bcr_d, 
			  0, 1, ny, ybc, bcb_d, bct_d,  
			  0, 1, nz, zbc, bcf_d, bce_d, 
			  lambda, f_d);
	)

	hipMemcpy(f, f_d, sizeof(double)*sz, hipMemcpyDeviceToHost);
	hipFree(f_d);
	if ( xbc == 3 || xbc == 4 )
		hipFree(bcl_d);
	if ( xbc == 3 || xbc == 2 )
		hipFree(bcr_d);
	if ( ybc == 3 || ybc == 4 )
		hipFree(bcb_d);
	if ( ybc == 3 || ybc == 2 )
		hipFree(bct_d);
	if ( zbc == 3 || zbc == 4 )
		hipFree(bcf_d);
	if ( zbc == 3 || zbc == 2 )
		hipFree(bce_d);

//	printf("***********************************\n");

	double maxim = 0;
	for (k=0; k<=nz; k++)
	{
		for (j=0; j<=ny; j++)
		{
			for (i=0; i<=nx; i++)
			{
				double dif = f[k*(nx+1)*(ny+1)+j*(nx+1)+i] - ufunc(i*dx,j*dy,k*dz);
#ifdef DEBUG
				printf("%12.8f", dif);
#endif
				dif = fabs(dif);
				if (maxim < dif)
					maxim = dif;
#ifdef DEBUG
				if (i%9 == 8)
					printf("\n");
#endif
			}
		}
#ifdef DEBUG
		printf("\n");
#endif
	}

	printf("max diff : %.14f\n", maxim);

	delete [] f;
	if ( xbc == 3 || xbc == 4 )
		free(bcl);
	if ( xbc == 3 || xbc == 2 )
		free(bcr);
	if ( ybc == 3 || ybc == 4 )
		free(bcb);
	if ( ybc == 3 || ybc == 2 )
		free(bct);
	if ( zbc == 3 || zbc == 4 )
		free(bcf);
	if ( zbc == 3 || zbc == 2 )
		free(bce);

	return 0;
}
